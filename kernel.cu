#include "hip/hip_runtime.h"
#include "kernel.h"
#include <stdio.h>

__global__ void mc_daoc_kernel( // Down and out call
    float *d_s,
    float T,
    float K,
    float B,
    float S0,
    float sigma,
    float mu,
    float r,
    float dt,
    float *d_normals,
    unsigned N_STEPS,
    unsigned N_PATHS)
{
    const unsigned tid = threadIdx.x;
    const unsigned bid = blockIdx.x;
    const unsigned bsz = blockDim.x;
    int s_idx = tid + bid * bsz;
    int n_idx = tid + bid * bsz;
    float s_curr = S0;
    if (s_idx < N_PATHS)
    {
        int n = 0;
        do
        {
            s_curr = s_curr + mu * s_curr * dt + sigma * s_curr * d_normals[n_idx];
            n_idx++;
            n++;
        } while (n < N_STEPS && s_curr > B);
        double payoff = (s_curr > K ? s_curr - K : 0.0);
        __syncthreads();
        d_s[s_idx] = exp(-r * T) * payoff;
    }
}

__global__ void mc_uaop_kernel( // Up and out put
    float *d_s,
    float T,
    float K,
    float B,
    float S0,
    float sigma,
    float mu,
    float r,
    float dt,
    float *d_normals,
    unsigned N_STEPS,
    unsigned N_PATHS)
{
    const unsigned tid = threadIdx.x;
    const unsigned bid = blockIdx.x;
    const unsigned bsz = blockDim.x;
    int s_idx = tid + bid * bsz;
    int n_idx = tid + bid * bsz;
    float s_curr = S0;
    if (s_idx < N_PATHS)
    {
        int n = 0;
        do
        {
            s_curr = s_curr + mu * s_curr * dt + sigma * s_curr * d_normals[n_idx];
            n_idx++;
            n++;
        } while (n < N_STEPS && s_curr < B);
        double payoff = (s_curr < K ? K - s_curr : 0.0);
        __syncthreads();
        d_s[s_idx] = exp(-r * T) * payoff;
    }
}

__global__ void mc_uaic_kernel( // Up and in call
    float *d_s,
    float T,
    float K,
    float B,
    float S0,
    float sigma,
    float mu,
    float r,
    float dt,
    float *d_normals,
    unsigned N_STEPS,
    unsigned N_PATHS)
{
    const unsigned tid = threadIdx.x;
    const unsigned bid = blockIdx.x;
    const unsigned bsz = blockDim.x;
    int s_idx = tid + bid * bsz;
    int n_idx = tid + bid * bsz;
    float s_curr = S0;
    if (s_idx < N_PATHS)
    {
        int n = 0;
        bool b_crossed = false;
        while (n < N_STEPS)
        {
            s_curr = s_curr + mu * s_curr * dt + sigma * s_curr * d_normals[n_idx];
            n_idx++;
            n++;
            if (s_curr > B)
                b_crossed = true;
        }
        double payoff = (b_crossed && s_curr > K ? s_curr - K : 0.0);
        __syncthreads();
        d_s[s_idx] = exp(-r * T) * payoff;
    }
}

__global__ void mc_daip_kernel( // Down and in put
    float *d_s,
    float T,
    float K,
    float B,
    float S0,
    float sigma,
    float mu,
    float r,
    float dt,
    float *d_normals,
    unsigned N_STEPS,
    unsigned N_PATHS)
{
    const unsigned tid = threadIdx.x;
    const unsigned bid = blockIdx.x;
    const unsigned bsz = blockDim.x;
    int s_idx = tid + bid * bsz;
    int n_idx = tid + bid * bsz;
    float s_curr = S0;
    if (s_idx < N_PATHS)
    {
        int n = 0;
        bool b_crossed = false;
        while (n < N_STEPS)
        {
            s_curr = s_curr + mu * s_curr * dt + sigma * s_curr * d_normals[n_idx];
            n_idx++;
            n++;
            if (s_curr < B)
                b_crossed = true;
        }
        double payoff = (b_crossed && s_curr < K ? K - s_curr : 0.0);
        __syncthreads();
        d_s[s_idx] = exp(-r * T) * payoff;
    }
}

void mc_daoc_call(
    float *d_s,
    float T,
    float K,
    float B,
    float S0,
    float sigma,
    float mu,
    float r,
    float dt,
    float *d_normals,
    unsigned N_STEPS,
    unsigned N_PATHS)
{
    if (B > S0)
    {
        printf("error: B > S0.\n");
        return;
    }
    const unsigned BLOCK_SIZE = 1024;
    const unsigned GRID_SIZE = ceil(float(N_PATHS) / float(BLOCK_SIZE));
    mc_daoc_kernel<<<GRID_SIZE, BLOCK_SIZE>>>(
        d_s, T, K, B, S0, sigma, mu, r, dt, d_normals, N_STEPS, N_PATHS);
}

void mc_uaop_call(
    float *d_s,
    float T,
    float K,
    float B,
    float S0,
    float sigma,
    float mu,
    float r,
    float dt,
    float *d_normals,
    unsigned N_STEPS,
    unsigned N_PATHS)
{
    if (B < S0)
    {
        printf("error: B < S0.\n");
        return;
    }
    const unsigned BLOCK_SIZE = 1024;
    const unsigned GRID_SIZE = ceil(float(N_PATHS) / float(BLOCK_SIZE));
    mc_uaop_kernel<<<GRID_SIZE, BLOCK_SIZE>>>(
        d_s, T, K, B, S0, sigma, mu, r, dt, d_normals, N_STEPS, N_PATHS);
}

void mc_uaic_call(
    float *d_s,
    float T,
    float K,
    float B,
    float S0,
    float sigma,
    float mu,
    float r,
    float dt,
    float *d_normals,
    unsigned N_STEPS,
    unsigned N_PATHS)
{
    if (B < S0)
    {
        printf("error: B < S0.\n");
        return;
    }
    const unsigned BLOCK_SIZE = 1024;
    const unsigned GRID_SIZE = ceil(float(N_PATHS) / float(BLOCK_SIZE));
    mc_uaic_kernel<<<GRID_SIZE, BLOCK_SIZE>>>(
        d_s, T, K, B, S0, sigma, mu, r, dt, d_normals, N_STEPS, N_PATHS);
}

void mc_daip_call(
    float *d_s,
    float T,
    float K,
    float B,
    float S0,
    float sigma,
    float mu,
    float r,
    float dt,
    float *d_normals,
    unsigned N_STEPS,
    unsigned N_PATHS)
{
    if (B > S0)
    {
        printf("error: B > S0.\n");
        return;
    }
    const unsigned BLOCK_SIZE = 1024;
    const unsigned GRID_SIZE = ceil(float(N_PATHS) / float(BLOCK_SIZE));
    mc_daip_kernel<<<GRID_SIZE, BLOCK_SIZE>>>(
        d_s, T, K, B, S0, sigma, mu, r, dt, d_normals, N_STEPS, N_PATHS);
}